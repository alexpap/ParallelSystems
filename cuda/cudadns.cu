#include "hip/hip_runtime.h"
/********************************************************************************/
/*DNS Algorithm									*/
/*Input	parameter	N dimesion of square matrices A,B,C			*/
/*Output		Computation time					*/
/*Compile		nvcc -o cudadns cudadns.c				*/
/*Usage			./cudadns <N>						*/
/********************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void dns(float *A,float *B,float *C,int N) 
{
	extern __shared__ float sum[];

	sum[threadIdx.x]=A[blockIdx.x*N+threadIdx.x]*B[threadIdx.x*N+blockIdx.y]; // multiplication 

	__syncthreads();

	for(unsigned int s=N/2; s>0; s>>=1) 					// reduction
	{
        	if (threadIdx.x < s) 
        	{
        	    sum[threadIdx.x] += sum[threadIdx.x + s];
        	}
        	__syncthreads();
    	}
	if ( !threadIdx.x ) C[blockIdx.x*N+blockIdx.y]=sum[threadIdx.x];	// result
}

int main(int argc, char* argv[])
{

	float *A, *B, *C;							// A,B,C matrices@host
	float *devA,*devB,*devC;						// A,B,C matrices @device 
	int i,j,N=0;								//i,j: counters , N:dims of A,B,C
	float elapsedTime;							//timers
	hipEvent_t start, stop;
	
	if ( argc != 2 || (N=atoi(argv[1])) < 1 )				//check cmd line args
	{
		printf("Please provide legal args\n");
		return -1;
	}
	(void)srand(time(NULL));				//seed random generator with the value of time in seconds since the Epoch

	A = (float*)malloc( N*N*sizeof(float) );				//Allocate host mem
	B = (float*)malloc( N*N*sizeof(float) );
	C = (float*)malloc( N*N*sizeof(float) );

	for( i=0; i<N ;i++)							//generate values for matrices A,B
		for( j=0; j<N; j++)
		{
			A[i*N+j]=(float)rand()/(float)RAND_MAX;	
			B[i*N+j]=(float)rand()/(float)RAND_MAX;
			C[i*N+j]=0.0;						//initialize matrix C
		}
	
	hipMalloc((void**)&devA,N*N*sizeof(float));					//Allocate  dev mem 
	hipMalloc((void**)&devB,N*N*sizeof(float));	
	hipMalloc((void**)&devC,N*N*sizeof(float));	
	hipMemcpy((void*)devA,A,N*N*sizeof(float),hipMemcpyHostToDevice);		// Copy A,B,C to device 
	hipMemcpy((void*)devB,B,N*N*sizeof(float),hipMemcpyHostToDevice);		
	hipMemcpy((void*)devC,C,N*N*sizeof(float),hipMemcpyHostToDevice);  	

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);						// start event	
	
	dns<<<dim3(N,N),N,N*sizeof(float)>>>(devA,devB,devC,N);		//run dns kernel with N*N blocks of N threads and a table sizeof N

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{ 
		printf("Error: %s\n", hipGetErrorString(err));
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipFree(devA);hipFree(devB);hipFree(devC);				//deallocate device mem
		free(A);free(B);free(C);						//deallocate host mem
		return -1;
	}

	hipEventRecord(stop, 0);						// Stop event
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop); 			// get elapsed time
	hipMemcpy(C, devC, N*N*sizeof(float), hipMemcpyDeviceToHost);		// get matrix C back from device
	hipFree(devA);hipFree(devB);hipFree(devC);				//deallocate device mem
	free(A);free(B);free(C);						//deallocate host mem
	
	printf("A, B, C 	<-- %d x %d matrices\n",N,N);					//output				
	printf("Computation time  : %f\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}
